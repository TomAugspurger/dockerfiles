
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloKernel() {
    printf("Hello world from GPU!\n");
}

void hello() {
    helloKernel<<<1,1>>>();
    hipDeviceSynchronize();
}
